#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <iostream>

using namespace cv;
using namespace std;

__global__ void MakeGrey(uchar* gpu_color_img, uchar* gpu_grey_img, int columns)
{
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    int gpu_color_pixel = (row * columns + col) * 3;
    gpu_grey_img[row * columns + col] = gpu_color_img[gpu_color_pixel] * 0.11 \
        + gpu_color_img[gpu_color_pixel + 1] * 0.59 + gpu_color_img[gpu_color_pixel + 2] * 0.3;
}

hipError_t CopyAndExecute(const Mat& color_img, const Mat& grey_img, int rows, int columns)
{
    uchar* gpu_color_img;
    uchar* gpu_grey_img;
    size_t color_img_size = color_img.rows * color_img.cols * sizeof(Vec3b);
    size_t grey_img_size = grey_img.rows * grey_img.cols * sizeof(uchar);
    cout << "size of color image: " << color_img_size << endl;
    cout << "size of grey img: " << grey_img_size << endl;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&gpu_color_img, color_img_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpu_grey_img, grey_img_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(gpu_color_img, color_img.data, color_img_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(gpu_grey_img, grey_img.data, grey_img_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    MakeGrey <<<1200,1000>>>(gpu_color_img, gpu_grey_img, columns);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(grey_img.data, gpu_grey_img, grey_img_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
        hipFree(gpu_color_img);
        hipFree(gpu_grey_img);
    
    return cudaStatus;
}

int main()
{
    //Print out NVIDIA device data
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cout << "Device Number: " << i << endl;
        cout << "Device name: " << prop.name << endl;
        cout << "Memory Clock Rate (KHz): " << prop.memoryClockRate << endl;
        cout << "Memory Bus Width (bits): " << prop.memoryBusWidth << endl;
        cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
        cout << "Warp Size: " << prop.warpSize << endl;
        cout << "Max grid size x, y, z: "
            << prop.maxGridSize[0] << ", "   
            << prop.maxGridSize[1] << ", "
            << prop.maxGridSize[2]
            << endl;
        cout << "Max threads x, y, z: " 
            << prop.maxThreadsDim[0] << ", "
            << prop.maxThreadsDim[1] << ", "
            << prop.maxThreadsDim[2]
            << endl;
    }

    //get data from color picture and create grey picture
    string image_path = samples::findFile("sunflower.jpg");
    Mat color_img = imread(image_path, IMREAD_COLOR);
    int rows = color_img.rows;
    int cols = color_img.cols;
    Mat grey_img = Mat::zeros(rows, cols, CV_8UC1);
    if (color_img.empty())
    {
        cout << "Could not read the image: " << image_path << std::endl;
        return 1;
    }

    hipError_t cudaStatus = CopyAndExecute(color_img, grey_img, rows, cols);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "take_input failed!");
        return 1;
    }
    
    /*
    for (int y = 0; y < rows; y++)
    {
        for (int x = 0; x < cols; x++)
        {
            Vec3b color_pixel = color_img.at<Vec3b>(Point(x, y));
            Scalar grey_pixel = grey_img.at<uchar>(Point(x, y));
            grey_pixel = (color_pixel[0] * 0.11 + color_pixel[1] * 0.59 + color_pixel[2] * 0.3);

            grey_img.at<uchar>(Point(x, y)) = grey_pixel.val[0];
        }
    }
    */


    imshow("Display Window", grey_img);
    imwrite("sunflower_greyscale.jpg", grey_img);
    waitKey(0);


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
