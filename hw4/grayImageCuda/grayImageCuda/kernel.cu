#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <iostream>

using namespace cv;
using namespace std;

__global__ void MakeGrey(uchar* gpu_color_img, uchar* gpu_grey_img, size_t grey_img_size)
{
    //get position of grey pixel in current CUDA thread
    int pixel = blockIdx.x * blockDim.x + threadIdx.x;
    //there are 3 color pixels per grey pixel
    int gpu_color_pixel = pixel * 3;
    //get the three values from the color picture and assign the corresponding grey value to the grey picture
    gpu_grey_img[pixel] = gpu_color_img[gpu_color_pixel] * 0.11 \
        + gpu_color_img[gpu_color_pixel + 1] * 0.59 + gpu_color_img[gpu_color_pixel + 2] * 0.3;
}

void FreeMemory(uchar* gpu_color_img, uchar* gpu_grey_img)
{
    hipFree(gpu_color_img);
    hipFree(gpu_grey_img);
}

hipError_t CopyAndExecute(const Mat& color_img, const Mat& grey_img, int rows, int columns)
{
    //create variables for use
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipError_t cudaStatus;
    uchar* gpu_color_img;
    uchar* gpu_grey_img;
    float milliseconds = 0;
    size_t color_img_size = color_img.rows * color_img.cols * sizeof(Vec3b);
    size_t grey_img_size = grey_img.rows * grey_img.cols;
    cout << "size of color image: " << color_img_size << endl;
    cout << "size of grey img: " << grey_img_size << endl;

    //allocate memory for the color and grey image on the GPU, then copy them over to device memory
    cudaStatus = hipMalloc((void**)&gpu_color_img, color_img_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed at color image!");
        FreeMemory(gpu_color_img, gpu_grey_img);
    }

    cudaStatus = hipMalloc((void**)&gpu_grey_img, grey_img_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed at grey image!");
        FreeMemory(gpu_color_img, gpu_grey_img);
    }

    cudaStatus = hipMemcpy(gpu_color_img, color_img.data, color_img_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed at color image!");
        FreeMemory(gpu_color_img, gpu_grey_img);
    }

    cudaStatus = hipMemcpy(gpu_grey_img, grey_img.data, grey_img_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed at grey image!");
        FreeMemory(gpu_color_img, gpu_grey_img);
    }
    
    //To get many data points, use a nested for loop with a variety of parameters
    int thread_numbers[6] = {32, 64, 128, 256, 512, 1024};
    for (int i = 0; i < 6; i++)
    {
        //ensure that we allocate the correct number of threads 
        int block_number = grey_img_size / thread_numbers[i];
        hipEventRecord(start);
        MakeGrey << <block_number, thread_numbers[i] >> > (gpu_color_img, gpu_grey_img, grey_img_size);
        hipEventRecord(stop);

        //record time with parameters
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        cout << "With " << thread_numbers[i] << " threads "
            << "and "<< block_number << " blocks "
            << "it took " << milliseconds << "milliseconds" << endl;      
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        FreeMemory(gpu_color_img, gpu_grey_img);
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        FreeMemory(gpu_color_img, gpu_grey_img);
    }

    cudaStatus = hipMemcpy(grey_img.data, gpu_grey_img, grey_img_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to host failed!");
        FreeMemory(gpu_color_img, gpu_grey_img);
    }
    return cudaStatus;
}

int main()
 {
    //Print out NVIDIA device data
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cout << "Device Number: " << i << endl;
        cout << "Device name: " << prop.name << endl;
        cout << "Memory Clock Rate (KHz): " << prop.memoryClockRate << endl;
        cout << "Memory Bus Width (bits): " << prop.memoryBusWidth << endl;
        cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
        cout << "Warp Size: " << prop.warpSize << endl;
        cout << "Max grid size x, y, z: "
            << prop.maxGridSize[0] << ", "   
            << prop.maxGridSize[1] << ", "
            << prop.maxGridSize[2]
            << endl;
        cout << "Max threads x, y, z: " 
            << prop.maxThreadsDim[0] << ", "
            << prop.maxThreadsDim[1] << ", "
            << prop.maxThreadsDim[2]
            << endl;
    }

    //try with a variety of image sizes
    string images[3] = { "outside-small", "outside-medium", "outside-large"};
    for (int i = 0; i < 3; i++)
    {
        cout << "Converting " << images[i] << endl;
        //use opencv to find the picture and create empty image of the same size
        string image_path = samples::findFile(images[i] + ".jpg");
        Mat color_img = imread(image_path, IMREAD_COLOR);
        int rows = color_img.rows;
        int cols = color_img.cols;
        Mat grey_img = Mat::zeros(rows, cols, CV_8UC1);
        if (color_img.empty())
        {
            cout << "Could not read the image: " << image_path << std::endl;
            return 1;
        }

        hipError_t cudaStatus = CopyAndExecute(color_img, grey_img, rows, cols);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "take_input failed!");
            return 1;
        }

        //save to file system
        imwrite(images[i] + "-greyscale" + ".jpg", grey_img);

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }
        cout << endl;
    }

    return 0;
}
