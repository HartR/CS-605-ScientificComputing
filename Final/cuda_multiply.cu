#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
 
using namespace std;

 __global__ void __multiply__ (double* a, double* b, double* c, int matrix_a_height, int matrix_a_width_matrix_b_height, int matrix_b_width, int offset)
 {
     
      
     int i = blockIdx.y * blockDim.y + threadIdx.y; 
     int j = blockIdx.x * blockDim.x + threadIdx.x;
     //printf ("ONE: %d is i, %d is j, width is %d, height is%d\n", i, j, matrix_b_width, matrix_a_height); 

     if( j <= offset && j < matrix_b_width * matrix_a_height) 
     {
         printf ("TWO: %d is j, offset is%d\n", j, offset); 
         for(int k = 0; k < matrix_a_width_matrix_b_height; k++) 
         {
               c[i * matrix_b_width + j] += a[i * matrix_a_width_matrix_b_height + k] * b[k * matrix_b_width + j];
               //printf("\n is %d, a is %f, b is %f", i, a[i * matrix_a_width_matrix_b_height + i], b[i * matrix_b_width + j]);
         }
         //printf("c[%i] is %f\n", i * matrix_b_width + j, c[i * matrix_b_width + j]);
         //printf("\matrix_a_width_matrix_b_height At location %d, in c, assigned value %f, sum is %f, value of a is %f, val of b is %f", i * matrix_b_width + j + offset, c[i * matrix_b_width + j + offset], a[i], b[i]);    
     }
     else if ( j > offset && j < matrix_b_width * matrix_a_height)
     {
          printf ("ONE: %d is j, offset is%d\n", j, offset); 

          for(int k = 0; k < matrix_a_width_matrix_b_height; k++) 
          {
                c[i * matrix_b_width + j] += a[i * matrix_a_width_matrix_b_height + k] * b[k * matrix_b_width + j];
                //printf("\n is %d, a is %f, b is %f", i, a[i * matrix_a_width_matrix_b_height + i], b[i * matrix_b_width + j]);
          }
     }
     
     /*
     if(i ==0 && j==0)
     {
          printf("\n a in cuda, with offset %d \n", offset);
          for (int i = 0; i < matrix_a_height * matrix_a_width_matrix_b_height; i++)
               printf("%d: %f, ", i, a[i]);
          printf("\n");

          printf("\n b in cuda, with offset %d \n", offset);
          for (int i = 0; i < matrix_a_width_matrix_b_height * matrix_b_width; i++)
               printf("%d: %f, ", i, b[i]);
          printf("\n");
     }*/
     



     
     /*int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     if (pixel < half_length)
     {
          printf("\nIn matrix b, current value in result: %f, value at %d: %f, ", result[pixel], pixel, b[pixel]);
          result[pixel] = b[pixel+offset];
     }
     printf("\matrix_a_width_matrix_b_height\matrix_a_width_matrix_b_height");*/
 }

 


void MatrixMultiplyCuda(double* mat_a, double* mat_b, double* mat_result, int matrix_a_height, int matrix_a_width_matrix_b_height, int matrix_b_width, int host_id)
{
     hipError_t cudaStatus;
     double* mat_a_device;
     double* mat_b_device;
     double* mat_result_device;

     //figure out ideal thread/block numbers
     //I'matrix_a_height using 256 threads, because we found that to be optimal from assignment 4
     int thread_number = 256;
     int block_number = 1;
     int mat_result_length = matrix_a_height*matrix_b_width;
     if(mat_result_length < thread_number)
     {
          thread_number = mat_result_length;
     }
     else if (mat_result_length > thread_number)
     {
          //get the ceiling of the division
          block_number = (mat_result_length + thread_number - 1)/thread_number;
     }
     
     //
     int offset = host_id * (mat_result_length)/2;


     hipMalloc((void**)&mat_a_device, sizeof(double)*matrix_a_height*matrix_a_width_matrix_b_height);
     hipMalloc((void**)&mat_b_device, sizeof(double)*matrix_a_width_matrix_b_height*matrix_b_width);
     hipMalloc((void**)&mat_result_device, sizeof(double)*mat_result_length);
     hipMemcpy(mat_a_device, mat_a, sizeof(double)*matrix_a_height*matrix_a_width_matrix_b_height, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, sizeof(double)*matrix_a_width_matrix_b_height*matrix_b_width, hipMemcpyHostToDevice);
     hipMemcpy(mat_result_device, mat_result, sizeof(double)*mat_result_length, hipMemcpyHostToDevice);



     __multiply__ <<<1, 256>>> (mat_a_device, mat_b_device, mat_result_device, matrix_a_height, matrix_a_width_matrix_b_height, matrix_b_width, offset);
     hipMemcpy(mat_result, mat_result_device, sizeof(double)*mat_result_length, hipMemcpyDeviceToHost);

     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);
     hipFree(mat_result_device);


     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
