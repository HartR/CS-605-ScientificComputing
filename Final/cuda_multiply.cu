#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#define BLOCK_SIZE 16
 
using namespace std;

 __global__ void __multiply__ (double* a, double* b, double* c, int matrix_a_height, int matrix_a_width_matrix_b_height, int matrix_b_width, int offset)
 {
     
      
     int i = blockIdx.y * blockDim.y + threadIdx.y; 
     int j = blockIdx.x * blockDim.x + threadIdx.x;

     if( j < matrix_b_width && i < matrix_a_height) 
     {
          
         for(int k = 0; k < matrix_a_width_matrix_b_height; k++) 
         {
               c[i * matrix_b_width + j] += a[i * matrix_a_width_matrix_b_height + matrix_b_width] * b[matrix_b_width * matrix_b_width + j];
               //printf("\ni is %d, a is %f, b is %f", i, a[i * matrix_a_width_matrix_b_height + i], b[i * matrix_b_width + j]);
         }
         //printf("\matrix_a_width_matrix_b_height At location %d, in c, assigned value %f, sum is %f, value of a is %f, val of b is %f", i * matrix_b_width + j + offset, c[i * matrix_b_width + j + offset], a[i], b[i]);    
     }
     if(i ==0 && j==0)
     {
          printf("\n a in cuda, with offset %d \n", offset);
          for (int i = 0; i < matrix_a_height * matrix_a_width_matrix_b_height; i++)
               printf("%d: %f, ", i, a[i]);
          printf("\n");

          printf("\n b in cuda, with offset %d \n", offset);
          for (int i = 0; i < matrix_a_width_matrix_b_height * matrix_b_width; i++)
               printf("%d: %f, ", i, b[i]);
          printf("\n");
     }



     
     /*int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     if (pixel < half_length)
     {
          printf("\nIn matrix b, current value in result: %f, value at %d: %f, ", result[pixel], pixel, b[pixel]);
          result[pixel] = b[pixel+offset];
     }
     printf("\matrix_a_width_matrix_b_height\matrix_a_width_matrix_b_height");*/
 }

 


void MatrixMultiplyCuda(double* mat_a, double* mat_b, double* mat_result, int matrix_a_height, int matrix_a_width_matrix_b_height, int matrix_b_width, int host_id)
{
     hipError_t cudaStatus;
     double* mat_a_device;
     double* mat_b_device;
     double* mat_result_device;

     //figure out ideal thread/block numbers
     //I'matrix_a_height using 256 threads, because we found that to be optimal from assignment 4
     /*int thread_number = 256;
     int block_number = 1;
     int array_length = matrix_a_height*matrix_b_width;
     if(array_length < thread_number)
     {
          thread_number = SIZE;
     }
     else if (SIZE > thread_number)
     {
          //get the ceiling of the division
          block_number = (array_length + thread_number - 1)/thread_number;
     }*/
     int offset = host_id * (matrix_a_height*matrix_b_width)/2;
     printf("\n a in buda before, with offset %d \n", offset);
     for (int i = 0; i < matrix_a_height * matrix_a_width_matrix_b_height; i++)
          printf("%d: %f, ", i, mat_a[i]);
     printf("\n");
     //thread_number*block_number == array_length/2
     hipMalloc((void**)&mat_a_device, sizeof(double)*matrix_a_height*matrix_a_width_matrix_b_height);
     hipMalloc((void**)&mat_b_device, sizeof(double)*matrix_a_width_matrix_b_height*matrix_b_width);
     hipMalloc((void**)&mat_result_device, sizeof(double)*matrix_a_height*matrix_b_width);
     hipMemcpy(mat_a_device, mat_a, sizeof(double)*matrix_a_height*matrix_a_width_matrix_b_height, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, sizeof(double)*matrix_a_width_matrix_b_height*matrix_b_width, hipMemcpyHostToDevice);
     hipMemcpy(mat_result_device, mat_result, sizeof(double)*matrix_a_height*matrix_b_width, hipMemcpyHostToDevice);
     //PrintMatrix(mat_result, sqrt(array_length), host_id);

     unsigned int grid_rows = (matrix_a_height + BLOCK_SIZE - 1) / BLOCK_SIZE;
     unsigned int grid_cols = (matrix_b_width + BLOCK_SIZE - 1) / BLOCK_SIZE;
     dim3 dimGrid(grid_cols, grid_rows);
     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

     __multiply__ <<<5, 6>>> (mat_a_device, mat_b_device, mat_result_device, matrix_a_height, matrix_a_width_matrix_b_height, matrix_b_width, offset);
     hipMemcpy(mat_result, mat_result_device, sizeof(double)*matrix_a_height*matrix_b_width, hipMemcpyDeviceToHost);

     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);
     hipFree(mat_result_device);


     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
