#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""


 
using namespace std;

 __global__ void __multiply__ (double* a, double* b, double* result)
 {
     int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     //printf("Value at %d: %f, ", pixel, a[pixel]);
     result[pixel] = b[pixel];

 }

 void PrintMatrix(double* matrix, int N, int p)
{
     printf("\nPrinting in CUDA on node %d \n", p);
    for (int row = 0; row < N; row++)
    {
        for (int column = 0; column < N; column++)
            printf("%f ", matrix[N * row + column]);
        printf("\n");
    }  
}
 

void MatrixMultiplyCuda(double* mat_a, double* mat_b, double* mat_result, int array_length)
{
     hipError_t cudaStatus;
     /* ... Load CPU data into GPU buffers  */
     double* mat_a_device;
     double* mat_b_device;
     double* mat_result_device;
     size_t matrix_size = array_length*sizeof(double);
     //I'm using 256 threads, because we found that to be optimal from assignment 4
     int thread_number = 256;
     int block_number;

     if(array_length > thread_number)
     {
          thread_number = array_length;
          block_number = 1;
     }
     else
     {
          block_number = array_length/thread_number;
     }
     //need to allocate result matrix
     hipMalloc((void**)&mat_a_device, matrix_size);
     hipMalloc((void**)&mat_b_device, matrix_size);
     hipMalloc((void**)&mat_result_device, matrix_size);
     hipMemcpy(mat_a_device, mat_a, matrix_size, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, matrix_size, hipMemcpyHostToDevice);
     __multiply__ <<<block_number, thread_number>>> (mat_a_device, mat_b_device, mat_result_device);
     hipMemcpy(mat_result, mat_result_device, matrix_size, hipMemcpyDeviceToHost);


     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);
     hipFree(mat_result_device);


     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
