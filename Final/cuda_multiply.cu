#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>


 
using namespace std;

 __global__ void __multiply__ (double* a, double* b, double* result, int offset, int half_length)
 {
     int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     if (pixel < half_length)
     {
          printf("\nIn matrix b, current value in result: %f, value at %d: %f, ", result[pixel], pixel, b[pixel]);
          result[pixel] = b[pixel+offset];
     }
     printf("\n\n");
 }

 /*
 void PrintMatrix(double* matrix, int N, int p)
{
     printf("\nPrinting in CUDA on host %d \n", p);
    for (int row = 0; row < N; row++)
    {
        for (int column = 0; column < N; column++)
            printf("%f ", matrix[N * row + column]);
        printf("\n");
    }  
}
 */

void MatrixMultiplyCuda(int host_id)
{
     hipError_t cudaStatus;
     double* mat_a_device;
     double* mat_b_device;
     double* mat_result_device;
     size_t matrix_size = SIZE*sizeof(double);

     //figure out ideal thread/block numbers
     //I'm using 256 threads, because we found that to be optimal from assignment 4
     int thread_number = 256;
     int block_number = 1;
     if(SIZE < thread_number)
     {
          thread_number = SIZE;
     }
     else if (SIZE > thread_number)
     {
          //get the ceiling of the division
          block_number = (SIZE + thread_number - 1)/thread_number;
     }
     int offset = host_id * (HALF);

     //thread_number*block_number == HALF
     hipMalloc((void**)&mat_a_device, matrix_size);
     hipMalloc((void**)&mat_b_device, matrix_size);
     hipMalloc((void**)&mat_result_device, matrix_size/2);
     hipMemcpy(mat_a_device, matrix_a, matrix_size, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, matrix_b, matrix_size, hipMemcpyHostToDevice);
     doubl* mat_result;
     if(host_id == 0)
          mat_result = matrix_result_1;
     else
          mat_result = matrix_result_2;
     
     hipMemcpy(mat_result_device, mat_result, matrix_size/2, hipMemcpyHostToDevice);

     //PrintMatrix(mat_result, sqrt(SIZE), host_id);

     __multiply__ <<<block_number, thread_number>>> (mat_a_device, mat_b_device, mat_result_device, offset, HALF);
     hipMemcpy(mat_result, mat_result_device, matrix_size/2, hipMemcpyDeviceToHost);


     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);
     hipFree(mat_result_device);


     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
