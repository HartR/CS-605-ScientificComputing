#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""


 
using namespace std;

 __global__ void __multiply__ (double* a, double* b)
 {
     int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     printf("Value at %d: %f, ", pixel, a[pixel]);

 }

 void PrintMatrix(double* matrix, int N)
{
     printf("\nPrinting in CUDA\n");
    for (int row = 0; row < N; row++)
    {
        for (int column = 0; column < N; column++)
            printf("%f ", matrix[N * row + column]);
        printf("\n");
    }  
}
 

void MatrixMultiplyCuda(double* mat_a, double* mat_b, int array_length)
{
     PrintMatrix(mat_a, array_length);
     hipError_t cudaStatus;
     /* ... Load CPU data into GPU buffers  */
     double* mat_a_device;
     double* mat_b_device;
     //need to allocate result matrix
     hipMalloc((void**)&mat_a_device, array_length*sizeof(double));
     hipMalloc((void**)&mat_b_device, array_length*sizeof(double));
     hipMemcpy(mat_a_device, mat_a, array_length*sizeof(double), hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, array_length*sizeof(double), hipMemcpyHostToDevice);

     __multiply__ <<<2, 2>>> (mat_a_device, mat_b_device);


     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);

     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
