#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#define BLOCK_SIZE 16
 
using namespace std;

 __global__ void __multiply__ (double* a, double* b, double* c, int matrix_1_height, int matrix_1_width_matrix_2_height, int matrix_2_width, int offset)
 {
     int i = blockIdx.y * blockDim.y + threadIdx.y; 
     int j = blockIdx.x * blockDim.x + threadIdx.x;
     int sum = 0;
     if( j < matrix_2_width && i < matrix_1_height) 
     {
         for(int matrix_2_width = 0; matrix_2_width < matrix_1_width_matrix_2_height; matrix_2_width++) 
         {
               c[i * matrix_2_width + j] += a[i * matrix_1_width_matrix_2_height + matrix_2_width] * b[matrix_2_width * matrix_2_width + j];
               printf("\ni is %d, a is %f, b is %f", i, a[i * matrix_1_width_matrix_2_height + i], b[i * matrix_2_width + j]);
         }
         printf("sum is %f\n", sum);
         c[i * matrix_2_width + j] = sum;
         printf("\n\n");
         //printf("\matrix_1_width_matrix_2_height At location %d, in c, assigned value %f, sum is %f, value of a is %f, val of b is %f", i * matrix_2_width + j + offset, c[i * matrix_2_width + j + offset], a[i], b[i]);

     }
     /*int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     if (pixel < half_length)
     {
          printf("\nIn matrix b, current value in result: %f, value at %d: %f, ", result[pixel], pixel, b[pixel]);
          result[pixel] = b[pixel+offset];
     }
     printf("\matrix_1_width_matrix_2_height\matrix_1_width_matrix_2_height");*/
 }

 /*
 void PrintMatrix(double* matrix, int matrix_1_width_matrix_2_height, int p)
{
     printf("\nPrinting in CUDA on host %d \matrix_1_width_matrix_2_height", p);
    for (int i = 0; i < matrix_1_width_matrix_2_height; i++)
    {
        for (int column = 0; column < matrix_1_width_matrix_2_height; column++)
            printf("%f ", matrix[matrix_1_width_matrix_2_height * i + column]);
        printf("\matrix_1_width_matrix_2_height");
    }  
}
 */

void MatrixMultiplyCuda(double* mat_a, double* mat_b, double* mat_result, int matrix_1_height, int matrix_1_width_matrix_2_height, int matrix_2_width, int host_id)
{
     hipError_t cudaStatus;
     double* mat_a_device;
     double* mat_b_device;
     double* mat_result_device;

     //figure out ideal thread/block numbers
     //I'matrix_1_height using 256 threads, because we found that to be optimal from assignment 4
     int thread_number = 256;
     int block_number = 1;
     int array_length = matrix_1_height*matrix_2_width;
     if(array_length < thread_number)
     {
          thread_number = array_length;
     }
     else if (array_length > thread_number)
     {
          //get the ceiling of the division
          block_number = (array_length + thread_number - 1)/thread_number;
     }
     int offset = host_id * ((matrix_1_height*matrix_2_width)/2);

     //thread_number*block_number == array_length/2
     hipMalloc((void**)&mat_a_device, sizeof(double)*matrix_1_height*matrix_1_width_matrix_2_height);
     hipMalloc((void**)&mat_b_device, sizeof(double)*matrix_1_width_matrix_2_height*matrix_2_width);
     hipMalloc((void**)&mat_result_device, sizeof(double)*matrix_1_height*matrix_2_width);
     hipMemcpy(mat_a_device, mat_a, sizeof(int)*matrix_1_height*matrix_1_width_matrix_2_height, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, sizeof(int)*matrix_1_width_matrix_2_height*matrix_2_width, hipMemcpyHostToDevice);
     hipMemcpy(mat_result_device, mat_result, sizeof(double)*matrix_1_height*matrix_2_width, hipMemcpyHostToDevice);
     //PrintMatrix(mat_result, sqrt(array_length), host_id);

     unsigned int grid_rows = (matrix_1_height + BLOCK_SIZE - 1) / BLOCK_SIZE;
     unsigned int grid_cols = (matrix_2_width + BLOCK_SIZE - 1) / BLOCK_SIZE;
     dim3 dimGrid(grid_cols, grid_rows);
     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

     __multiply__ <<<dimGrid, dimBlock>>> (mat_a_device, mat_b_device, mat_result_device, matrix_1_height, matrix_1_width_matrix_2_height, matrix_2_width, offset);
     hipMemcpy(mat_result, mat_result_device, sizeof(double)*matrix_1_height*matrix_2_width, hipMemcpyDeviceToHost);

     mat_result[0] = 9999;
     mat_result[1] = 7777;
     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);
     hipFree(mat_result_device);


     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
