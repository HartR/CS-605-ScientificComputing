#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
 
 __global__ void __multiply__ (double* a, double* b)
 {
          printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
 }
 

void call_me_maybe(double* mat_a, double* mat_b, int array_length)
{
     hipError_t cudaStatus;
     /* ... Load CPU data into GPU buffers  */
     double* mat_a_device;
     double* mat_b_device;
     //need to allocate result matrix
     hipMalloc((void**)&mat_a_device, array_length*sizeof(double));
     hipMalloc((void**)&mat_b_device, array_length*sizeof(double));
     hipMemcpy(mat_a_device, &mat_a, array_length*sizeof(double), hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, &mat_b, array_length*sizeof(double), hipMemcpyHostToDevice);
     __multiply__ <<<5, 5>>> (mat_a_device, mat_b_device);


     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);

     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
