#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
 
 __global__ void __multiply__ (double* a, double* b)
 {
          fprintf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
 }
 
void FreeMemory(double* mat_a_device, double* mat_b_device)
{
     hipFree(mat_a_device);
     hipFree(mat_b_device);
}
void call_me_maybe(size_t a_size, size_t b_size, double* mat_a, double* mat_b)
{
     hipError_t cudaStatus;
     /* ... Load CPU data into GPU buffers  */
     double* mat_a_device;
     double* mat_b_device;
     //need to allocate result matrix
     hipMalloc((void**)&mat_a_device, a_size);
     hipMalloc((void**)&mat_b_device, b_size);
     hipMemcpy(mat_a_device, mat_a, a_size, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, b_size, hipMemcpyHostToDevice);
     __multiply__ <<<5, 5>>> (mat_a_device, mat_b_device);


     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
          FreeMemory(mat_a_device, mat_b_device);
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
          FreeMemory(mat_a_device, mat_b_device);
     }

     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
