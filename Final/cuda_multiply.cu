#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
 
 __global__ void __multiply__ (double* a, double* b)
 {
          printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
 }
 
void call_me_maybe(int blocksize, size_t a_size, size_t b_size, double* mat_a, double* mat_b)
{
     /* ... Load CPU data into GPU buffers  */
    double* mat_a_device;
    double* mat_b_device;
    //need to allocate result matrix
    hipMalloc((void**)&mat_a_device, a_size);
    hipMalloc((void**)&mat_b_device, b_size);
    //hipMemcpy(mat_a_device, mat_a, a_size, hipMemcpyHostToDevice);
    //hipMemcpy(mat_b_device, mat_b, b_size, hipMemcpyHostToDevice);
     //__multiply__ <<<5, 5>>> (mat_a_device, mat_b_device);
     hipFree(mat_a_device);
     hipFree(mat_b_device);
 
     /* ... Transfer data from GPU to CPU */
}
