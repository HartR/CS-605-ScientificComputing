#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""


 
using namespace std;

 __global__ void __multiply__ (double* a, double* b)
 {
     int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     printf("Value at %d: %f, ", pixel, a[pixel]);
     b[pixel] = 0;

 }

 void PrintMatrix(double* matrix, int N, int p)
{
     printf("\nPrinting in CUDA on node %d \n", p);
    for (int row = 0; row < N; row++)
    {
        for (int column = 0; column < N; column++)
            printf("%f ", matrix[N * row + column]);
        printf("\n");
    }  
}
 

void MatrixMultiplyCuda(double* mat_a, double* mat_b, int array_length, int p)
{
     hipError_t cudaStatus;
     /* ... Load CPU data into GPU buffers  */
     double* mat_a_device;
     double* mat_b_device;
     //need to allocate result matrix
     hipMalloc((void**)&mat_a_device, array_length*sizeof(double));
     hipMalloc((void**)&mat_b_device, array_length*sizeof(double));
     hipMemcpy(mat_a_device, mat_a, array_length*sizeof(double), hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, array_length*sizeof(double), hipMemcpyHostToDevice);

     __multiply__ <<<2, 2>>> (mat_a_device, mat_b_device);


     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);

     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
