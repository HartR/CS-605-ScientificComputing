#include "hip/hip_runtime.h"
/* multiply.cu */
#include <hip/hip_runtime.h>
#include "mycuda.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#define BLOCK_SIZE 16
 
using namespace std;

 __global__ void __multiply__ (double* a, double* b, double* c, int m, int n, int k, int offset)
 {
     int row = blockIdx.y * blockDim.y + threadIdx.y; 
     int col = blockIdx.x * blockDim.x + threadIdx.x;
     int sum = 0;
     if( col < k && row < m) 
     {
         for(int i = 0; i < n/2; i++) 
         {
             sum += a[row * n + i] * b[i * k + col];
             printf("\ni is %d, a is %f, b is %f", i, a[row * n + i], b[i * k + col]);
         }
         printf("sum is %f\n", sum);
         c[row * k + col] = sum;
         printf("\n\n");
         //printf("\n At location %d, in c, assigned value %f, sum is %f, value of a is %f, val of b is %f", row * k + col + offset, c[row * k + col + offset], a[i], b[i]);

     }
     printf("\n\n");
     /*int pixel = blockIdx.x * blockDim.x + threadIdx.x;
     if (pixel < half_length)
     {
          printf("\nIn matrix b, current value in result: %f, value at %d: %f, ", result[pixel], pixel, b[pixel]);
          result[pixel] = b[pixel+offset];
     }
     printf("\n\n");*/
 }

 /*
 void PrintMatrix(double* matrix, int N, int p)
{
     printf("\nPrinting in CUDA on host %d \n", p);
    for (int row = 0; row < N; row++)
    {
        for (int column = 0; column < N; column++)
            printf("%f ", matrix[N * row + column]);
        printf("\n");
    }  
}
 */

void MatrixMultiplyCuda(double* mat_a, double* mat_b, double* mat_result, int m, int n, int k, int host_id)
{
     hipError_t cudaStatus;
     double* mat_a_device;
     double* mat_b_device;
     double* mat_result_device;

     //figure out ideal thread/block numbers
     //I'm using 256 threads, because we found that to be optimal from assignment 4
     int thread_number = 256;
     int block_number = 1;
     int array_length = m*k;
     if(array_length < thread_number)
     {
          thread_number = array_length;
     }
     else if (array_length > thread_number)
     {
          //get the ceiling of the division
          block_number = (array_length + thread_number - 1)/thread_number;
     }
     int offset = host_id * ((m*k)/2);

     //thread_number*block_number == array_length/2
     hipMalloc((void**)&mat_a_device, sizeof(double)*m*n);
     hipMalloc((void**)&mat_b_device, sizeof(double)*n*k);
     hipMalloc((void**)&mat_result_device, sizeof(double)*m*k);
     hipMemcpy(mat_a_device, mat_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
     hipMemcpy(mat_b_device, mat_b, sizeof(int)*n*k, hipMemcpyHostToDevice);
     hipMemcpy(mat_result_device, mat_result, sizeof(double)*m*k, hipMemcpyHostToDevice);
     //PrintMatrix(mat_result, sqrt(array_length), host_id);

     unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
     unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
     dim3 dimGrid(grid_cols, grid_rows);
     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

     __multiply__ <<<dimGrid, dimBlock>>> (mat_a_device, mat_b_device, mat_result_device, m, n, k, offset);
     hipMemcpy(mat_result, mat_result_device, sizeof(double)*m*k, hipMemcpyDeviceToHost);

     mat_result[0] = 9999;
     mat_result[1] = 7777;
     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
     }

     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
     }

     hipFree(mat_a_device);
     hipFree(mat_b_device);
     hipFree(mat_result_device);


     cudaStatus = hipDeviceReset();
     if (cudaStatus != hipSuccess) {
          fprintf(stderr, "hipDeviceReset failed!");
     }     /* ... Transfer data from GPU to CPU */
}
